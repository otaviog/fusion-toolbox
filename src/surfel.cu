#include "hip/hip_runtime.h"
#include "surfel.hpp"

#include "accessor.hpp"
#include "camera.hpp"
#include "error.hpp"
#include "kernel.hpp"

namespace fiontb {
namespace {
template <Device dev, typename scalar_t>
struct ComputeConfidencesKernel {
  const KCamera<dev, scalar_t> kcam;
  const scalar_t constant_weight;
  const scalar_t weight;
  const scalar_t max_center_distance;
  typename Accessor<dev, scalar_t, 2>::T confidences;

  ComputeConfidencesKernel(const torch::Tensor &kcam, scalar_t weight,
                           scalar_t max_center_distance,
                           torch::Tensor confidences)
      : kcam(kcam),
        constant_weight(2.0 * pow(0.6, 2)),
        weight(weight),
        max_center_distance(max_center_distance),
        confidences(Accessor<dev, scalar_t, 2>::Get(confidences)) {}

  FTB_DEVICE_HOST void operator()(int row, int col) {
    const Vector<scalar_t, 2> camera_center(kcam.get_center());
    scalar_t confidence =
        (Vector<scalar_t, 2>(col, row) - camera_center).norm();
    confidence = confidence / max_center_distance;
    confidence = exp(-(confidence * confidence) / constant_weight) * weight;

    confidences[row][col] = confidence;
  }
};

}  // namespace

void SurfelOp::ComputeConfidences(const torch::Tensor &kcam, float weight,
                                  float max_center_distance,
                                  torch::Tensor confidences) {
  const auto ref_device = confidences.device();
  FTB_CHECK_DEVICE(ref_device, kcam);

  const auto ref_type = confidences.scalar_type();
  if (ref_device.is_cuda()) {
    AT_DISPATCH_FLOATING_TYPES(ref_type, "ComputeConfidences", [&] {
      ComputeConfidencesKernel<kCUDA, scalar_t> kernel(
          kcam, weight, max_center_distance, confidences);
      Launch2DKernelCUDA(kernel, confidences.size(1), confidences.size(0));
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES(ref_type, "ComputeConfidences", [&] {
      ComputeConfidencesKernel<kCPU, scalar_t> kernel(
          kcam, weight, max_center_distance, confidences);
      Launch2DKernelCPU(kernel, confidences.size(1), confidences.size(0));
    });
  }
}

namespace {
template <Device dev, typename scalar_t>
struct ComputeRadiiKernel {
  const typename Accessor<dev, scalar_t, 1>::T depths;
  const typename Accessor<dev, scalar_t, 1>::T normals_z;
  typename Accessor<dev, scalar_t, 1>::T radii;
  scalar_t focal_len;

  ComputeRadiiKernel(const torch::Tensor &kcam_, const torch::Tensor &depths,
                     const torch::Tensor &normals_z, torch::Tensor radii)
      : depths(Accessor<dev, scalar_t, 1>::Get(depths)),
        normals_z(Accessor<dev, scalar_t, 1>::Get(normals_z)),
        radii(Accessor<dev, scalar_t, 1>::Get(radii)) {
    const KCamera<kCPU, scalar_t> kcam(kcam_);
    focal_len =
        (abs(kcam.matrix[0][0]) + abs(kcam.matrix[1][1])) * 0.5;
  }

  FTB_DEVICE_HOST void operator()(int idx) {
    const scalar_t _1_sqrt_2 = 0.7071067811865475;
    const scalar_t radius = _1_sqrt_2 * (depths[idx] / focal_len);

    radii[idx] = min(radius / abs(normals_z[idx]), 2 * radius);
  }
};
}  // namespace

void SurfelOp::ComputeRadii(const torch::Tensor &kcam,
                            const torch::Tensor &depths,
                            const torch::Tensor &normals_z,
                            torch::Tensor radii) {
  const auto ref_device = normals_z.device();
  FTB_CHECK_DEVICE(ref_device, kcam);
  FTB_CHECK_DEVICE(ref_device, radii);

  const auto ref_type = normals_z.scalar_type();
  if (ref_device.is_cuda()) {
    AT_DISPATCH_FLOATING_TYPES(ref_type, "ComputeRadii", [&] {
      ComputeRadiiKernel<kCUDA, scalar_t> kernel(kcam, depths, normals_z,
                                                 radii);
      Launch1DKernelCUDA(kernel, normals_z.size(0));
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES(ref_type, "ComputeRadii", [&] {
      ComputeRadiiKernel<kCPU, scalar_t> kernel(kcam, depths, normals_z, radii);
      Launch1DKernelCPU(kernel, normals_z.size(0));
    });
  }
}

}  // namespace fiontb
