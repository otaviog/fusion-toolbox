#include "hip/hip_runtime.h"
#include "icpodometry.hpp"

#include "cuda_utils.hpp"
#include "math.hpp"

namespace fiontb {
struct KCamera {
  KCamera(torch::Tensor kcam_matrix)
      : kcam_matrix(
            kcam_matrix.packed_accessor<float, 2, torch::RestrictPtrTraits,
                                        size_t>()) {}
  __device__ Eigen::Vector2i project(const Eigen::Vector3f point) {
    const float img_x =
	  kcam_matrix[0][0] * point[0] / point[2] + kcam_matrix[0][2];
    const float img_y =
	  kcam_matrix[1][1] * point[1] / point[2] + kcam_matrix[1][2];

    return Eigen::Vector2i(round(img_x),						   
						   round(img_y));
  }

  const PackedAccessor<float, 2> kcam_matrix;
};

struct RTCamera {
  RTCamera(torch::Tensor rt_matrix)
      : rt_matrix(rt_matrix.packed_accessor<float, 2, torch::RestrictPtrTraits,
                                            size_t>()) {}

  __device__ Eigen::Vector3f transform(const Eigen::Vector3f point) const {
    const auto mtx = rt_matrix;
    const float px = mtx[0][0] * point[0] + mtx[0][1] * point[1] +
                     mtx[0][2] * point[2] + mtx[0][3];
    const float py = mtx[1][0] * point[0] + mtx[1][1] * point[1] +
                     mtx[1][2] * point[2] + mtx[1][3];
    const float pz = mtx[2][0] * point[0] + mtx[2][1] * point[1] +
                     mtx[2][2] * point[2] + mtx[2][3];

    return Eigen::Vector3f(px, py, pz);
  }
  const PackedAccessor<float, 2> rt_matrix;
};

struct JacobianKernel {
  const PackedAccessor<float, 3> points0;
  const PackedAccessor<float, 3> normals0;
  const PackedAccessor<uint8_t, 2> mask0;
  const PackedAccessor<float, 2> points1;
  const PackedAccessor<uint8_t, 1> mask1;
  KCamera kcam;
  RTCamera prev_rt_cam;

  PackedAccessor<float, 2> jacobian;
  PackedAccessor<float, 1> residual;

  JacobianKernel(const PackedAccessor<float, 3> points0,
                 const PackedAccessor<float, 3> normals0,
                 const PackedAccessor<uint8_t, 2> mask0,
                 const PackedAccessor<float, 2> points1,
                 const PackedAccessor<uint8_t, 1> mask1, KCamera kcam,
                 RTCamera prev_rt_cam, PackedAccessor<float, 2> jacobian,
                 PackedAccessor<float, 1> residual)
      : points0(points0),
        normals0(normals0),
        mask0(mask0),
        points1(points1),
        mask1(mask1),
        kcam(kcam),
        prev_rt_cam(prev_rt_cam),
        jacobian(jacobian),
        residual(residual) {}

  __device__ void EstimateJacobian() {
    const int ri = blockIdx.x * blockDim.x + threadIdx.x;
    if (ri >= points1.size(0)) return;

	jacobian[ri][0] = 0.0f;
    jacobian[ri][1] = 0.0f;
    jacobian[ri][2] = 0.0f;
    jacobian[ri][3] = 0.0f;
    jacobian[ri][4] = 0.0f;
    jacobian[ri][5] = 0.0f;
    residual[ri] = 0.0f;
	
	if (mask1[ri] == 0) return;

    const int width = points0.size(1);
    const int height = points0.size(0);

    const Eigen::Vector3f p1_on_prev =
	  prev_rt_cam.transform(to_vec3<float>(points1[ri]));
	Eigen::Vector2i p1_proj = kcam.project(p1_on_prev);
    if (p1_proj[0] < 0 || p1_proj[0] >= width || p1_proj[1] < 0 ||
        p1_proj[1] >= height)
      return;
	if (mask0[p1_proj[1]][p1_proj[0]] == 0) return;
    const Eigen::Vector3f point0(to_vec3<float>(points0[p1_proj[1]][p1_proj[0]]));
    const Eigen::Vector3f normal0(
        to_vec3<float>(normals0[p1_proj[1]][p1_proj[0]]));

    jacobian[ri][0] = normal0[0];
    jacobian[ri][1] = normal0[1];
    jacobian[ri][2] = normal0[2];

    const Eigen::Vector3f rot_twist = p1_on_prev.cross(normal0);
    jacobian[ri][3] = rot_twist[0];
    jacobian[ri][4] = rot_twist[1];
    jacobian[ri][5] = rot_twist[2];

    residual[ri] = (point0 - p1_on_prev).dot(normal0);
  }
};

__global__ void EstimateJacobian_gpu_kernel(JacobianKernel kernel) {
  kernel.EstimateJacobian();
}

void EstimateJacobian_gpu(const torch::Tensor points0,
                          const torch::Tensor normals0,
						  const torch::Tensor mask0,
                          const torch::Tensor points1,
						  const torch::Tensor mask1,
						  const torch::Tensor kcam,
                          const torch::Tensor rt_cam, torch::Tensor jacobian,
                          torch::Tensor residual) {
  JacobianKernel estm_kern(
      points0.packed_accessor<float, 3, torch::RestrictPtrTraits, size_t>(),
      normals0.packed_accessor<float, 3, torch::RestrictPtrTraits, size_t>(),
	  mask0.packed_accessor<uint8_t, 2, torch::RestrictPtrTraits, size_t>(),
      points1.packed_accessor<float, 2, torch::RestrictPtrTraits, size_t>(),
	  mask1.packed_accessor<uint8_t, 1, torch::RestrictPtrTraits, size_t>(),
      KCamera(kcam), RTCamera(rt_cam),
      jacobian.packed_accessor<float, 2, torch::RestrictPtrTraits, size_t>(),
      residual.packed_accessor<float, 1, torch::RestrictPtrTraits, size_t>());

  CudaKernelDims kl = Get1DKernelDims(points1.size(0));
  EstimateJacobian_gpu_kernel<<<kl.grid, kl.block>>>(estm_kern);
  CudaCheck();
  CudaSafeCall(hipDeviceSynchronize());
}

#if 0
static torch::Tensor estimate_update(torch::Tensor points0, torch::Tensor normals0,
                                torch::Tensor points1, torch::Tensor kcam,
                                torch::Tensor transform,
                                torch::Tensor curr_params,
                                torch::Tensor curr_residual,
                                torch::Tensor prev_residual) {

  auto jacobian = EstimateJacobian_gpu(points0, normals0, points1, kcam, transform, curr_params,
                                       curr_residual, prev_residual);
  torch::Tensor JtJ = jacobian.transpose(1, 0).matmul(jacobian);
  auto inv_JtJ = JtJ.inverse();
  auto Jr = jacobian.matmul(curr_residual);

  torch::Tensor param_update = inv_JtJ.matmul(Jr);
  return param_update;
}
#endif

ICPOdometry::ICPOdometry(std::vector<float> scales, std::vector<int> num_iters)
    : scales_(scales), num_iters_(num_iters) {}

torch::Tensor ICPOdometry::Estimate(torch::Tensor points0,
                                    torch::Tensor normals0,
                                    torch::Tensor points1, torch::Tensor kcam,
                                    torch::Tensor transform) {
  torch::Tensor params = torch::zeros(
      {6}, torch::TensorOptions(torch::kFloat).device(torch::kCUDA, 0));
#if 0
  torch::Tensor residual0 =
      torch::zeros({points1.size(0)},
                   torch::TensorOptions(torch::kFloat).device(torch::kCUDA, 0));
  torch::Tensor residual1 = residual0.clone();

  for (int scale_idx = 0; scale_idx < scales_.size(); ++scale_idx) {
    const float scale = scales_[scale_idx];  // TODO: rescale image
    const int num_iters = num_iters_[scale_idx];

    for (int i = 0; i < num_iters; ++i) {
      torch::Tensor param_update =
          estimate_update(points0, normals0, points1, kcam, transform, params,
                          residual0, residual1);
      params = params + param_update;
      std::swap(residual0, residual1);
    }
  }
#endif
  return params;
}
}  // namespace fiontb