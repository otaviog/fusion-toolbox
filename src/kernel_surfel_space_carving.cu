#include "hip/hip_runtime.h"
#include <torch/torch.h>

#include "cuda_utils.hpp"
#include "math.hpp"

namespace fiontb {

namespace {

struct Framebuffer {
  Framebuffer(const PackedAccessor<float, 3> position,
              const PackedAccessor<int32_t, 3> index)
      : position(position), index(index) {}

  __device__ __host__ int width() const { return position.size(1); }
  __device__ __host__ int height() const { return position.size(0); }
  __device__ bool empty(int row, int col) const {
    return index[row][col][1] == 0;
  }

  const PackedAccessor<float, 3> position;
  const PackedAccessor<int32_t, 3> index;
};

const int MAX_VIOLANTIONS = 4;

__global__ void CarveSpace_gpu_kernel(Framebuffer stable_and_new,
                                      Framebuffer model,
                                      PackedAccessor<uint8_t, 1> free_mask,
                                      int search_size, float min_z_diff) {
  const int row = blockIdx.y * blockDim.y + threadIdx.y;
  const int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row >= model.height() || col >= model.width()) return;
  if (model.empty(row, col)) return;

  const float model_z = model.position[row][col][2];
  const int model_idx = model.index[row][col][0];

  //if (model.position[row][col][3] < 20) return;
  
  /**
   * Stable_And_New and model position framebuffers may differ in size.
   */
  const int wscale = model.width() / stable_and_new.width();
  const int hscale = model.height() / stable_and_new.height();

  int violantion_count = 0;

  const int start_row = max((row - search_size) / hscale, 0);
  const int end_row = min((row + search_size) / hscale, model.height() - 1);

  const int start_col = max((col - search_size) / wscale, 0);
  const int end_col = min((col + search_size) / wscale, model.width() - 1);

  for (int krow = start_row; krow <= end_row; ++krow) {
    for (int kcol = start_col; kcol <= end_col; ++kcol) {
      if (stable_and_new.empty(krow, kcol)) continue;      
      const float stable_z = stable_and_new.position[krow][kcol][2];
      
      if (stable_z - model_z > min_z_diff) {
        ++violantion_count;
      }
    }
  }

  if (violantion_count >= MAX_VIOLANTIONS) {    
    free_mask[model_idx] = 1;
  }
}
}  // namespace

void CarveSpace(const torch::Tensor stable_and_new_pos_fb,
                const torch::Tensor stable_and_new_idx_fb,
                const torch::Tensor model_pos_fb,
                const torch::Tensor model_idx_fb, torch::Tensor free_mask,
                int search_size, float min_z_diff) {  
  Framebuffer stable_and_new(
      stable_and_new_pos_fb
          .packed_accessor<float, 3, torch::RestrictPtrTraits, size_t>(),
      stable_and_new_idx_fb
          .packed_accessor<int32_t, 3, torch::RestrictPtrTraits, size_t>());

  Framebuffer model(
      model_pos_fb
          .packed_accessor<float, 3, torch::RestrictPtrTraits, size_t>(),
      model_idx_fb
          .packed_accessor<int32_t, 3, torch::RestrictPtrTraits, size_t>());

  const CudaKernelDims kern_dims = Get2DKernelDims(model.width(), model.height());
  CarveSpace_gpu_kernel<<<kern_dims.grid, kern_dims.block>>>(
      stable_and_new, model,
      free_mask.packed_accessor<uint8_t, 1, torch::RestrictPtrTraits, size_t>(),
      search_size, min_z_diff);
  CudaCheck();
  CudaSafeCall(hipDeviceSynchronize());
}
}  // namespace fiontb