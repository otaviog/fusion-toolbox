#include "hip/hip_runtime.h"
#include "surfel_fusion_common.hpp"

#include <torch/torch.h>

#include "camera.hpp"
#include "cuda_utils.hpp"
#include "eigen_common.hpp"
#include "error.hpp"
#include "kernel.hpp"
#include "math.hpp"

namespace fiontb {
namespace {

const int MAX_MERGE_VIOLATIONS = 1;

template <Device dev>
struct CleanKernel {
  SurfelModelAccessor<dev> model;
  const typename Accessor<dev, int64_t, 1>::T model_indices;
  const IndexMapAccessor<dev> model_indexmap;

  const KCamera<dev, float> kcam;
  const RTCamera<dev, float> inverse_rt;
  const Eigen::Matrix3f inverse_rt_normal;

  const int time;
  const int max_time_thresh;
  const int neighbor_size;
  const float stable_conf_thresh;

  typename Accessor<dev, bool, 1>::T remove_mask;

  CleanKernel(MappedSurfelModel model, torch::Tensor model_indices,
              const IndexMap &model_indexmap, const torch::Tensor &kcam,
              const torch::Tensor &inverse_rt,
              const Eigen::Matrix3f &inverse_rt_normal, int time,
              int max_time_thresh, int neighbor_size, float stable_conf_thresh,
              torch::Tensor remove_mask)
      : model(model),
        model_indices(Accessor<dev, int64_t, 1>::Get(model_indices)),
        model_indexmap(model_indexmap),
        kcam(kcam),
        inverse_rt(inverse_rt),
        inverse_rt_normal(inverse_rt_normal),
        time(time),
        max_time_thresh(max_time_thresh),
        neighbor_size(neighbor_size),
        stable_conf_thresh(stable_conf_thresh),
        remove_mask(Accessor<dev, bool, 1>::Get(remove_mask)) {}

  FTB_DEVICE_HOST void operator()(int idx) {
    int64_t model_idx = model_indices[idx];

    if (model.confidences[model_idx] < stable_conf_thresh &&
        (time - model.times[model_idx]) > max_time_thresh) {
      remove_mask[idx] = true;
      return;
    }

    const Eigen::Vector3f model_xyz = model.position(model_idx);
    const Eigen::Vector3f model_normal = model.normal(model_idx);

    const Eigen::Vector3f model_local_xyz = inverse_rt.Transform(model_xyz);
    const Eigen::Vector3f model_local_normal = inverse_rt_normal * model_normal;

    const float model_radius = model.radii[model_idx];
    const int model_time = model.times[model_idx];

    int u, v;
    kcam.Projecti(model_local_normal, u, v);

    const int width = model_indexmap.width();
    const int height = model_indexmap.height();
    if (u < 0 || u >= width || v < 0 || v >= height) return;

    if (model_indexmap.empty(v, u)) return;
    if (model_indexmap.confidence(v, u) < stable_conf_thresh) return;

    const int start_row = max(v - neighbor_size, 0);
    const int end_row = min(v + neighbor_size, height - 1);

    const int start_col = max(u - neighbor_size, 0);
    const int end_col = min(u + neighbor_size, width - 1);

    int merge_count = 0;
    int carve_count = 0;

    for (int krow = start_row; krow <= end_row; ++krow) {
      for (int kcol = start_col; kcol <= end_col; ++kcol) {
        if (model_indexmap.empty(krow, kcol)) continue;
        if (model_indexmap.confidence(krow, kcol) < stable_conf_thresh)
          continue;

        const Eigen::Vector3f im_xyz = model_indexmap.position(krow, kcol);
        const Eigen::Vector3f im_normal = model_indexmap.normal(krow, kcol);
        const float im_conf = model_indexmap.confidence(krow, kcol);

        if (im_conf > stable_conf_thresh &&
            im_xyz[2] > model_local_xyz[2] &
                im_xyz[2] - model_local_xyz[2] < 0.01 &&
            Eigen::Vector2f(im_xyz[0] - model_local_xyz[0],
                            im_xyz[1] - model_local_xyz[0])
                    .norm() < model_radius * 1.4) {
          ++merge_count;
        }

        if (model_time == time && im_conf > stable_conf_thresh &&
            im_xyz[2] > model_local_xyz[2] &&
            im_xyz[2] - model_local_xyz[2] > 0.01 &&
            abs(model_local_xyz[2]) > 0.85) {
          ++carve_count;
        }
      }
    }

    if (merge_count > 8 || carve_count > 4) {
      // Remove
      remove_mask[idx] = true;
    }
  }
};

}  // namespace

void SurfelFusionOp::Clean(MappedSurfelModel model, torch::Tensor model_indices,
                           const IndexMap &model_indexmap,
                           const torch::Tensor &kcam,
                           const torch::Tensor &world_to_cam, int time,
                           int max_time_thresh, int neighbor_size,
                           float stable_conf_thresh,
                           torch::Tensor remove_mask) {
  const auto ref_device = model_indexmap.get_device();
  model.CheckDevice(ref_device);
  model_indexmap.CheckDevice(ref_device);

  FTB_CHECK_DEVICE(ref_device, kcam);
  FTB_CHECK_DEVICE(ref_device, remove_mask);

  const auto world_to_cam_cpu = world_to_cam.cpu();
  Eigen::Matrix4f inverse_rt_cam(
      to_matrix<float, 4, 4>(world_to_cam_cpu.accessor<float, 2>()));
  Eigen::Matrix3f inverse_rt_normal(inverse_rt_cam.topLeftCorner(3, 3));
  inverse_rt_normal = inverse_rt_normal.inverse().transpose();
  if (ref_device.is_cuda()) {
    CleanKernel<kCUDA> kernel(model, model_indices, model_indexmap, kcam,
                              world_to_cam, inverse_rt_normal, time,
                              max_time_thresh, neighbor_size,
                              stable_conf_thresh, remove_mask);
    Launch1DKernelCUDA(kernel, model_indices.size(0));
  } else {
    CleanKernel<kCUDA> kernel(model, model_indices, model_indexmap, kcam,
                              world_to_cam, inverse_rt_normal, time,
                              max_time_thresh, neighbor_size,
                              stable_conf_thresh, remove_mask);
    Launch1DKernelCPU(kernel, model_indices.size(0));
  }
}
}  // namespace fiontb
