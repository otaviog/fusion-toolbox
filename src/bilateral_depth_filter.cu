#include "hip/hip_runtime.h"
#include "processing.hpp"

#include "accessor.hpp"
#include "cuda_utils.hpp"
#include "error.hpp"
#include "kernel.hpp"

#include <iostream>

using namespace std;

namespace slamtb {

template <Device dev, typename scalar_t>
struct BilateralDepthFilterKernel {
  const typename Accessor<dev, scalar_t, 2>::T input;
  const typename Accessor<dev, bool, 2>::T mask;
  typename Accessor<dev, scalar_t, 2>::T result;
  int half_width;
  float inv_sigma_color_sqr, inv_sigma_space_sqr, depth_scale;

  BilateralDepthFilterKernel(const torch::Tensor input,
                             const torch::Tensor mask, torch::Tensor result,
                             int half_width, float inv_sigma_color_sqr,
                             float inv_sigma_space_sqr, float depth_scale)
      : input(Accessor<dev, scalar_t, 2>::Get(input)),
        mask(Accessor<dev, bool, 2>::Get(mask)),
        result(Accessor<dev, scalar_t, 2>::Get(result)),
        half_width(half_width),
        inv_sigma_color_sqr(inv_sigma_color_sqr),
        inv_sigma_space_sqr(inv_sigma_space_sqr),
        depth_scale(depth_scale) {
  }

  STB_DEVICE_HOST void operator()(int row, int col) {
    result[row][col] = 0;
    if (mask[row][col] == 0) return;

    const float depth = input[row][col] * depth_scale;

    float color_sum = 0.0f;
    float weight_sum = 0.0f;
    const float inv_depth_scale = 1.0 / depth_scale;

    const int height = input.size(0);
    const int width = input.size(1);

    for (int y = -half_width; y <= half_width; ++y) {
      const int krow = row + y;
      if (krow < 0 || krow >= height) continue;

      for (int x = -half_width; x <= half_width; ++x) {
        const int kcol = col + x;
        if (kcol < 0 || kcol >= width) continue;
        if (mask[krow][kcol] == 0) continue;

        const float curr_depth = input[krow][kcol] * depth_scale;

        const float dx = col - kcol;
        const float dy = row - krow;
        const float space_sqr = dx * dx + dy * dy;

        const float dcolor = depth - curr_depth;
        const float color_sqr = dcolor * dcolor;

        const float weight = expf(-0.5f * (space_sqr * inv_sigma_space_sqr +
                                           color_sqr * inv_sigma_color_sqr));
        color_sum += curr_depth * weight;
        weight_sum += weight;
      }
    }

    if (weight_sum > 0.0f) {
      result[row][col] = scalar_t((color_sum / weight_sum) * inv_depth_scale);
    }
  }
};

torch::Tensor Processing::BilateralDepthFilter(
    const torch::Tensor &input, const torch::Tensor &mask, torch::Tensor result,
    int filter_width, float sigma_color, float sigma_space, float depth_scale) {
  const int width = input.size(1);
  const int height = input.size(0);

  const int half_width = filter_width / 2;
  const float inv_sigma_color_sqr = 1.0 / (sigma_color * sigma_color);
  const float inv_sigma_space_sqr = 1.0 / (sigma_space * sigma_space);

  STB_CHECK_DEVICE(input.device(), mask);
  STB_CHECK_DEVICE(input.device(), result);

  if (input.is_cuda()) {
    AT_DISPATCH_ALL_TYPES(
        input.scalar_type(), "BilateralFilterDepthImage_gpu", ([&] {
          BilateralDepthFilterKernel<kCUDA, scalar_t> kernel(
              input, mask, result, half_width, inv_sigma_color_sqr,
              inv_sigma_space_sqr, depth_scale);
          Launch2DKernelCUDA(kernel, input.size(1), input.size(0));
        }));
  } else {
    AT_DISPATCH_ALL_TYPES(
        input.scalar_type(), "BilateralFilterDepthImage_cpu", ([&] {
          BilateralDepthFilterKernel<kCPU, scalar_t> kernel(
              input, mask, result, half_width, inv_sigma_color_sqr,
              inv_sigma_space_sqr, depth_scale);
          Launch2DKernelCPU(kernel, input.size(1), input.size(0));
        }));
  }
  return result;
}

}  // namespace slamtb
