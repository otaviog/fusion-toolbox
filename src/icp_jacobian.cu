#include "hip/hip_runtime.h"
#include "icpodometry.hpp"

#include "accessor.hpp"
#include "camera.hpp"
#include "error.hpp"
#include "feature_map.hpp"
#include "kernel.hpp"
#include "pointgrid.hpp"

namespace fiontb {
namespace {

template <Device dev, typename scalar_t>
class PointGrid : public BasePointGrid<dev> {
 public:
  const typename Accessor<dev, scalar_t, 3>::T points;
  const typename Accessor<dev, scalar_t, 3>::T normals;

  PointGrid(const torch::Tensor &points, const torch::Tensor normals,
            const torch::Tensor &mask)
      : BasePointGrid<dev>(mask),
        points(Accessor<dev, scalar_t, 3>::Get(points)),
        normals(Accessor<dev, scalar_t, 3>::Get(normals)) {}
};

template <Device dev, typename scalar_t>
struct GeometricJacobianKernel {
  PointGrid<dev, scalar_t> tgt;
  const typename Accessor<dev, scalar_t, 2>::T src_points;
  const typename Accessor<dev, bool, 1>::T src_mask;
  KCamera<dev, scalar_t> kcam;
  RTCamera<dev, scalar_t> rt_cam;

  typename Accessor<dev, scalar_t, 2>::T jacobian;
  typename Accessor<dev, scalar_t, 1>::T residual;

  GeometricJacobianKernel(PointGrid<dev, scalar_t> tgt,
                          const torch::Tensor &src_points,
                          const torch::Tensor &src_mask,
                          KCamera<dev, scalar_t> kcam,
                          RTCamera<dev, scalar_t> rt_cam,
                          torch::Tensor jacobian, torch::Tensor residual)
      : tgt(tgt),
        src_points(Accessor<dev, scalar_t, 2>::Get(src_points)),
        src_mask(Accessor<dev, bool, 1>::Get(src_mask)),
        kcam(kcam),
        rt_cam(rt_cam),
        jacobian(Accessor<dev, scalar_t, 2>::Get(jacobian)),
        residual(Accessor<dev, scalar_t, 1>::Get(residual)) {}

  FTB_DEVICE_HOST void operator()(int ri) {
    jacobian[ri][0] = 0.0f;
    jacobian[ri][1] = 0.0f;
    jacobian[ri][2] = 0.0f;
    jacobian[ri][3] = 0.0f;
    jacobian[ri][4] = 0.0f;
    jacobian[ri][5] = 0.0f;
    residual[ri] = 0.0f;

    if (src_mask[ri] == 0) return;

    const int width = tgt.points.size(1);
    const int height = tgt.points.size(0);

    const Vector<scalar_t, 3> Tsrc_point =
        rt_cam.Transform(to_vec3<scalar_t>(src_points[ri]));
    Eigen::Vector2i src_uv = kcam.Project(Tsrc_point);
    if (src_uv[0] < 0 || src_uv[0] >= width || src_uv[1] < 0 ||
        src_uv[1] >= height)
      return;

    if (tgt.empty(src_uv[1], src_uv[0])) return;
    const Vector<scalar_t, 3> tgt_point(
        to_vec3<scalar_t>(tgt.points[src_uv[1]][src_uv[0]]));
    const Vector<scalar_t, 3> tgt_normal(
        to_vec3<scalar_t>(tgt.normals[src_uv[1]][src_uv[0]]));

    jacobian[ri][0] = tgt_normal[0];
    jacobian[ri][1] = tgt_normal[1];
    jacobian[ri][2] = tgt_normal[2];

    const Vector<scalar_t, 3> rot_twist = Tsrc_point.cross(tgt_normal);
    jacobian[ri][3] = rot_twist[0];
    jacobian[ri][4] = rot_twist[1];
    jacobian[ri][5] = rot_twist[2];

    residual[ri] = (tgt_point - Tsrc_point).dot(tgt_normal);
  }
};

}  // namespace

void ICPJacobian::EstimateGeometric(
    const torch::Tensor tgt_points, const torch::Tensor tgt_normals,
    const torch::Tensor tgt_mask, const torch::Tensor src_points,
    const torch::Tensor src_mask, const torch::Tensor kcam,
    const torch::Tensor rt_cam, torch::Tensor jacobian,
    torch::Tensor residual) {
  const auto reference_dev = src_points.device();

  FTB_CHECK_DEVICE(reference_dev, tgt_points);
  FTB_CHECK_DEVICE(reference_dev, tgt_normals);
  FTB_CHECK_DEVICE(reference_dev, tgt_mask);

  FTB_CHECK_DEVICE(reference_dev, src_mask);
  FTB_CHECK_DEVICE(reference_dev, kcam);
  FTB_CHECK_DEVICE(reference_dev, rt_cam);

  FTB_CHECK_DEVICE(reference_dev, jacobian);
  FTB_CHECK_DEVICE(reference_dev, residual);

  if (reference_dev.is_cuda()) {
    AT_DISPATCH_FLOATING_TYPES(
        src_points.scalar_type(), "EstimateICPJacobian", [&] {
          GeometricJacobianKernel<kCUDA, scalar_t> kernel(
              PointGrid<kCUDA, scalar_t>(tgt_points, tgt_normals, tgt_mask),
              src_points, src_mask, kcam, rt_cam, jacobian, residual);
          Launch1DKernelCUDA(kernel, src_points.size(0));
        });
  } else {
    AT_DISPATCH_FLOATING_TYPES(
        src_points.scalar_type(), "EstimateICPJacobian", [&] {
          GeometricJacobianKernel<kCPU, scalar_t> kernel(
              PointGrid<kCPU, scalar_t>(tgt_points, tgt_normals, tgt_mask),
              src_points, src_mask, kcam, rt_cam, jacobian, residual);
          Launch1DKernelCPU(kernel, src_points.size(0));
        });
  }
}

namespace {
template <Device dev, typename scalar_t>
FTB_DEVICE_HOST inline scalar_t EuclideanDistance(
    const BilinearInterp<dev, scalar_t> f1,
    const typename Accessor<dev, scalar_t, 2>::T f2, int f2_index) {
  scalar_t dist = scalar_t(0);
  for (int channel = 0; channel < f2.size(0); ++channel) {
    const scalar_t diff = f1.Get(channel) - f2[channel][f2_index];
    dist += diff * diff;
  }

  return sqrt(dist);
}

template <typename scalar_t>
FTB_DEVICE_HOST inline scalar_t Df1_EuclideanDistance(
    scalar_t f1_nth_val, scalar_t f2_nth_val, scalar_t inv_forward_result) {
  if (inv_forward_result > 0)
    return (f1_nth_val - f2_nth_val) * inv_forward_result;
  else
    return 0;
}

template <Device dev, typename scalar_t>
struct HybridJacobianKernel {
  PointGrid<dev, scalar_t> tgt;
  FeatureMap<dev, scalar_t> tgt_featmap;

  const typename Accessor<dev, scalar_t, 2>::T src_points;
  const typename Accessor<dev, scalar_t, 2>::T src_feats;
  const typename Accessor<dev, bool, 1>::T src_mask;

  KCamera<dev, scalar_t> kcam;
  RTCamera<dev, scalar_t> rt_cam;

  scalar_t geom_weight, feat_weight;

  typename Accessor<dev, scalar_t, 2>::T jacobian;
  typename Accessor<dev, scalar_t, 1>::T residual;

  HybridJacobianKernel(
      const PointGrid<dev, scalar_t> tgt, FeatureMap<dev, scalar_t> tgt_featmap,
      const torch::Tensor &src_points, const torch::Tensor &src_feats,
      const torch::Tensor &src_mask, KCamera<dev, scalar_t> kcam,
      RTCamera<dev, scalar_t> rt_cam, scalar_t geom_weight,
      scalar_t feat_weight, torch::Tensor jacobian, torch::Tensor residual)
      : tgt(tgt),
        tgt_featmap(tgt_featmap),
        src_points(Accessor<dev, scalar_t, 2>::Get(src_points)),
        src_feats(Accessor<dev, scalar_t, 2>::Get(src_feats)),
        src_mask(Accessor<dev, bool, 1>::Get(src_mask)),
        kcam(kcam),
        rt_cam(rt_cam),
        geom_weight(geom_weight),
        feat_weight(feat_weight),
        jacobian(Accessor<dev, scalar_t, 2>::Get(jacobian)),
        residual(Accessor<dev, scalar_t, 1>::Get(residual)) {}

  FTB_DEVICE_HOST void ComputeGeometricTerm(
      const Vector<scalar_t, 3> &Tsrc_point, int ui, int vi,
      scalar_t out_jacobian[6], scalar_t &out_residual) {
    const Vector<scalar_t, 3> tgt_point(to_vec3<scalar_t>(tgt.points[vi][ui]));
    const Vector<scalar_t, 3> tgt_normal(
        to_vec3<scalar_t>(tgt.normals[vi][ui]));

    out_jacobian[0] = tgt_normal[0];
    out_jacobian[1] = tgt_normal[1];
    out_jacobian[2] = tgt_normal[2];

    const Vector<scalar_t, 3> rot_twist = Tsrc_point.cross(tgt_normal);
    out_jacobian[3] = rot_twist[0];
    out_jacobian[4] = rot_twist[1];
    out_jacobian[5] = rot_twist[2];

    out_residual = tgt_normal.dot(tgt_point - Tsrc_point);
  }

#pragma nv_exec_check_disable
  FTB_DEVICE_HOST void ComputeFeatTerm(int ri,
                                       const Vector<scalar_t, 3> &Tsrc_point,
                                       scalar_t u, scalar_t v,
                                       scalar_t out_jacobian[6],
                                       scalar_t &out_residual) {
    BilinearInterp<dev, scalar_t> interp = tgt_featmap.GetBilinear(u, v);

    const scalar_t feat_residual = EuclideanDistance(interp, src_feats, ri);

    const scalar_t inv_feat_residual =
        (feat_residual > 0) ? scalar_t(1) / feat_residual : -1;

    const BilinearInterpGrad<dev, scalar_t> dx_interp(
        tgt_featmap.GetBilinearGrad(u, v));

    scalar_t d_euc_u = 0;
    scalar_t d_euc_v = 0;
    for (int channel = 0; channel < tgt_featmap.channel_size; ++channel) {
      const scalar_t df1_dist = Df1_EuclideanDistance(
          interp.Get(channel), src_feats[channel][ri], inv_feat_residual);

      scalar_t du, dv;
      dx_interp.Get(channel, du, dv);

      d_euc_u += df1_dist * du;
      d_euc_v += df1_dist * dv;
    }

	scalar_t j00_proj, j02_proj, j11_proj, j12_proj;
    kcam.Dx_Projection(Tsrc_point, j00_proj, j02_proj, j11_proj, j12_proj);
	
    Eigen::Matrix<scalar_t, 1, 3> pgrad;
    pgrad << d_euc_u * j00_proj, d_euc_v * j11_proj,
        d_euc_u * j02_proj + d_euc_v * j12_proj;
	
    Eigen::Matrix<scalar_t, 3, 3> K;
    K << kcam.matrix[0][0], kcam.matrix[0][1], kcam.matrix[0][2],
        kcam.matrix[1][0], kcam.matrix[1][1], kcam.matrix[1][2],
        kcam.matrix[2][0], kcam.matrix[2][1], kcam.matrix[2][2];

    Eigen::Matrix<scalar_t, 3, 6> J;
    // clang-format off
	J <<
	  1, 0, 0, 0, Tsrc_point[2], -Tsrc_point[1],
	  0, 1, 0, -Tsrc_point[2], 0, Tsrc_point[0],
	  0, 0, 1, Tsrc_point[1], -Tsrc_point[0], 0;
    // clang-format on

    //J = K * J;
    Eigen::Matrix<scalar_t, 1, 6> jacobian = pgrad * J;

    for (int k = 0; k < 6; ++k) out_jacobian[k] = jacobian(0, k);

    out_residual = feat_residual;
  }

  FTB_DEVICE_HOST void operator()(int ri) {
    jacobian[ri][0] = 0.0f;
    jacobian[ri][1] = 0.0f;
    jacobian[ri][2] = 0.0f;
    jacobian[ri][3] = 0.0f;
    jacobian[ri][4] = 0.0f;
    jacobian[ri][5] = 0.0f;
    residual[ri] = 0.0f;

    if (src_mask[ri] == 0) return;

    const int width = tgt.points.size(1);
    const int height = tgt.points.size(0);

    const Vector<scalar_t, 3> Tsrc_point =
        rt_cam.Transform(to_vec3<scalar_t>(src_points[ri]));

    scalar_t u, v;
    kcam.Project(Tsrc_point, u, v);

    const int ui = int(round(u));
    const int vi = int(round(v));
    if (ui < 0 || ui >= width || vi < 0 || vi >= height) return;
    if (tgt.empty(vi, ui)) return;

    scalar_t feat_jacobian[6], feat_residual;
    feat_jacobian[0] = feat_jacobian[1] = feat_jacobian[2] = feat_jacobian[3] =
        feat_jacobian[4] = feat_jacobian[5] = 0;
    feat_residual = 0;
    scalar_t geom_jacobian[6], geom_residual;

    if (ri == 228479) {
      geom_residual = 0;
    }
    ComputeFeatTerm(ri, Tsrc_point, u, v, feat_jacobian, feat_residual);
    ComputeGeometricTerm(Tsrc_point, ui, vi, geom_jacobian, geom_residual);
#pragma unroll
    for (int k = 0; k < 6; ++k) {
      jacobian[ri][k] =
          geom_jacobian[k] * geom_weight + feat_jacobian[k] * feat_weight;
    }

    residual[ri] = geom_residual * geom_weight + feat_residual * feat_weight;
  }
};

}  // namespace

void ICPJacobian::EstimateHybrid(
    const torch::Tensor tgt_points, const torch::Tensor tgt_normals,
    const torch::Tensor tgt_feats, const torch::Tensor tgt_mask,
    const torch::Tensor src_points, const torch::Tensor src_feats,
    const torch::Tensor src_mask, const torch::Tensor kcam,
    const torch::Tensor rt_cam, float geom_weight, float feat_weight,
    torch::Tensor jacobian, torch::Tensor residual) {
  const auto reference_dev = src_points.device();
  FTB_CHECK_DEVICE(reference_dev, tgt_points);
  FTB_CHECK_DEVICE(reference_dev, tgt_normals);
  FTB_CHECK_DEVICE(reference_dev, tgt_feats);
  FTB_CHECK_DEVICE(reference_dev, tgt_mask);

  FTB_CHECK_DEVICE(reference_dev, src_feats);
  FTB_CHECK_DEVICE(reference_dev, src_mask);

  FTB_CHECK_DEVICE(reference_dev, kcam);
  FTB_CHECK_DEVICE(reference_dev, rt_cam);
  FTB_CHECK_DEVICE(reference_dev, jacobian);
  FTB_CHECK_DEVICE(reference_dev, residual);

  if (reference_dev.is_cuda()) {
    AT_DISPATCH_FLOATING_TYPES(
        src_points.scalar_type(), "EstimateHybrid", ([&] {
          PointGrid<kCUDA, scalar_t> tgt(tgt_points, tgt_normals, tgt_mask);
          HybridJacobianKernel<kCUDA, scalar_t> kernel(
              tgt, FeatureMap<kCUDA, scalar_t>(tgt_feats), src_points,
              src_feats, src_mask, KCamera<kCUDA, scalar_t>(kcam),
              RTCamera<kCUDA, scalar_t>(rt_cam), geom_weight, feat_weight,
              jacobian, residual);
          Launch1DKernelCUDA(kernel, src_points.size(0));
        }));
  } else {
    AT_DISPATCH_FLOATING_TYPES(src_points.scalar_type(), "EstimateHybrid", [&] {
      PointGrid<kCPU, scalar_t> tgt(tgt_points, tgt_normals, tgt_mask);
      HybridJacobianKernel<kCPU, scalar_t> kernel(
          tgt, FeatureMap<kCPU, scalar_t>(tgt_feats), src_points, src_feats,
          src_mask, KCamera<kCPU, scalar_t>(kcam),
          RTCamera<kCPU, scalar_t>(rt_cam), geom_weight, feat_weight, jacobian,
          residual);
      Launch1DKernelCPU(kernel, src_points.size(0));
    });
  }
}
}  // namespace fiontb
