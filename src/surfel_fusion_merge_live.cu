#include "hip/hip_runtime.h"
#include "surfel_fusion_common.hpp"

#include "camera.hpp"
#include "kernel.hpp"
#include "math.hpp"

namespace fiontb {
namespace {

template <Device dev>
struct LiveMergeKernel {
  const IndexMapAccessor<dev> target_indexmap;
  const IndexMapAccessor<dev> live_indexmap;
  const typename Accessor<dev, float, 3>::T live_features;
  SurfelModelAccessor<dev> surfel_model;
  const RTCamera<dev, float> rt_cam;
  const Eigen::Matrix3f normal_transform_matrix;

  int scale, search_size;
  float max_normal_angle;

  typename Accessor<dev, int64_t, 2>::T new_surfel_map;

  LiveMergeKernel(const IndexMap &target_indexmap,
                  const IndexMap &live_indexmap,
                  const torch::Tensor &live_features,
                  MappedSurfelModel surfel_model, RTCamera<dev, float> rt_cam,
                  const Eigen::Matrix3f &normal_transform_matrix,
                  int search_size, float max_normal_angle,
                  torch::Tensor new_surfel_map)
      : target_indexmap(target_indexmap),
        live_indexmap(live_indexmap),
        live_features(Accessor<dev, float, 3>::Get(live_features)),
        surfel_model(surfel_model),
        rt_cam(rt_cam),
        normal_transform_matrix(normal_transform_matrix),
        search_size(search_size),
        max_normal_angle(max_normal_angle),
        new_surfel_map(Accessor<dev, int64_t, 2>::Get(new_surfel_map)) {
    scale =
        int(float(target_indexmap.get_height()) / live_indexmap.get_height());
    search_size = int(scale * search_size);
  }
#pragma nv_exec_check_disable
  FTB_DEVICE_HOST void operator()(int row, int col) {
    new_surfel_map[row][col] = -1;
    if (live_indexmap.empty(row, col)) return;

    const Vector<float, 3> live_pos(live_indexmap.position(row, col));

    const float lambda =
        sqrt(live_pos[0] * live_pos[0] + live_pos[1] * live_pos[1] + 1);
    const Vector<float, 3> ray(live_pos[0], live_pos[1], 1);

    const Vector<float, 3> live_normal(live_indexmap.normal(row, col));

    const int xstart = max(col * scale - search_size, 0);
    const int xend =
        min(col * scale + search_size, int(target_indexmap.width()) - 1);

    const int ystart = max(row * scale - search_size, 0);
    const int yend =
        min(row * scale + search_size, int(target_indexmap.height()) - 1);

    float best_dist = NumericLimits<dev, float>::infinity();
    int best = -1;

    for (int krow = ystart; krow <= yend; krow++) {
      for (int kcol = xstart; kcol <= xend; kcol++) {
        if (target_indexmap.empty(krow, kcol)) continue;

        const int current = target_indexmap.index(krow, kcol);

        const Vector<float, 3> model_pos = target_indexmap.position(krow, kcol);
        if (abs((model_pos[2] * lambda) - (live_pos[2] * lambda)) >= 0.05)
          continue;

        const float dist = ray.cross(model_pos).norm() / ray.norm();

        const Vector<float, 3> normal = target_indexmap.normal(krow, kcol);
        if (dist < best_dist &&
            (GetVectorsAngle(normal, live_normal) < .5  // max_normal_angle
             || abs(normal[2]) < 0.75f)) {
          best_dist = dist;
          best = current;
        }
      }
    }

    if (best >= 0) {
      const float live_conf = live_indexmap.confidence(row, col);
      const float model_conf = surfel_model.confidences[best];
      const float conf_total = live_conf + model_conf;

      const float live_radius = live_indexmap.radius(row, col);
      const float model_radius = surfel_model.radii[best];

      if (live_radius < (1.0 + 0.5) * model_radius) {
        const Vector<float, 3> live_world_pos = rt_cam.Transform(live_pos);
        surfel_model.set_position(best,
                                  (surfel_model.position(best) * model_conf +
                                   live_world_pos * live_conf) /
                                      conf_total);
        const Vector<float, 3> live_world_normal =
            normal_transform_matrix * live_normal;
        surfel_model.set_normal(best, (surfel_model.normal(best) * model_conf +
                                       live_world_normal * live_conf) /
                                          conf_total);

        const Vector<float, 3> live_color(live_indexmap.color(row, col));
        surfel_model.set_color(best, (surfel_model.color(best) * model_conf +
                                      live_color * live_conf) /
                                         conf_total);
        const int64_t feature_size =
            min(surfel_model.features.size(0), live_features.size(0));
        const int live_height = live_features.size(1);
        for (int64_t i = 0; i < feature_size; ++i) {
          const float model_feat_channel = surfel_model.features[i][best];
          // Indexmap comes up side down
          const float live_feat_channel =
              live_features[i][live_height - 1 - row][col];

#if 1
          surfel_model.features[i][best] = (model_feat_channel * model_conf +
                                            live_feat_channel * live_conf) /
                                           conf_total;
#else
          surfel_model.features[i][best] = live_feat_channel;
#endif
        }
      }
      surfel_model.confidences[best] = conf_total;
      surfel_model.times[best] = live_indexmap.time(row, col);
    } else {
      new_surfel_map[row][col] = live_indexmap.index(row, col);
    }
  }
};

Eigen::Matrix3f GetNormalTransformMatrix(const torch::Tensor rt_cam) {
  auto rt_cam_cpu = rt_cam.cpu();
  const torch::TensorAccessor<float, 2> acc = rt_cam_cpu.accessor<float, 2>();
  Eigen::Matrix3f mtx = Eigen::Matrix3f::Identity();

  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < 3; ++j) {
      mtx(i, j) = acc[i][j];
    }
  }

  return mtx.inverse().transpose();
}
}  // namespace

void SurfelFusionOp::MergeLive(const IndexMap &target_indexmap,
                               const IndexMap &live_indexmap,
                               const torch::Tensor &live_features,
                               MappedSurfelModel model,
                               const torch::Tensor &rt_cam, int search_size,
                               float max_normal_angle,
                               torch::Tensor new_surfels_map) {
  auto reference_dev = target_indexmap.get_device();

  live_indexmap.CheckDevice(reference_dev);
  target_indexmap.CheckDevice(reference_dev);
  model.CheckDevice(reference_dev);
  FTB_CHECK_DEVICE(reference_dev, rt_cam);

  Eigen::Matrix3f normal_transform_matrix(GetNormalTransformMatrix(rt_cam));
  if (reference_dev.is_cuda()) {
    LiveMergeKernel<kCUDA> kernel(target_indexmap, live_indexmap, live_features,
                                  model, RTCamera<kCUDA, float>(rt_cam),
                                  normal_transform_matrix, search_size,
                                  max_normal_angle, new_surfels_map);
    Launch2DKernelCUDA(kernel, live_indexmap.get_width(),
                       live_indexmap.get_height());
  } else {
    LiveMergeKernel<kCPU> kernel(target_indexmap, live_indexmap, live_features,
                                 model, RTCamera<kCPU, float>(rt_cam),
                                 normal_transform_matrix, search_size,
                                 max_normal_angle, new_surfels_map);
    Launch2DKernelCPU(kernel, live_indexmap.get_width(),
                      live_indexmap.get_height());
  }
}
}  // namespace fiontb
